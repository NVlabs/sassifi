/* Copyright (c) 2017, NVIDIA CORPORATION. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*    contributors may be used to endorse or promote products derived
*    from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <stdio.h>
#include <stdint.h>
#include <unistd.h>
#include <assert.h>
#include <string>
#include <map>
#include <fstream>
#include <sys/time.h>
#include <cupti.h>

#include <sassi/sassi-core.hpp>
#include <sassi/sassi-regs.hpp>
#include <sassi/sassi-memory.hpp>
#include "sassi/sassi-opcodes.h"
#include "sassi_intrinsics.h"
#include "sassi_dictionary.hpp"
#include "sassi_lazyallocator.hpp"

#include "error_injector.h"

std::map<std::string, int> knameCount;

std::ofstream ofs; 

#if TIMING
struct timeval start, end;
float mTotalTime = 0;
#endif

// This function will be called before every SASS instruction gets executed 
__device__ void sassi_before_handler(SASSIBeforeParams* bp, SASSIMemoryParams *mp, SASSIRegisterParams *rp) {
#if EMPTY_HANDLER
	return;
#endif

  if (bp->GetInstrWillExecute()) {
		profile_instructions(bp, mp, rp);
	} else {
		profile_will_not_execute_instructions();
	}

}

// This function will be exected before a kernel is launced
static void onKernelEntry(const CUpti_CallbackData *cbInfo) {
	reset_profiling_counters(); // reset profiling counters

#if TIMING 
	gettimeofday(&start, NULL);
#endif
} 

// This function will be exected after the kernel exits 
static void onKernelExit(const CUpti_CallbackData *cbInfo) {

	hipError_t * error = (hipError_t*) cbInfo->functionReturnValue; 
	if ( (*error) != hipSuccess ) {
		printf("Kernel Exit Error: %d", (*error));
	}

	// print per thread counters
	std::string kName = cbInfo->symbolName; // name of kernel
	if (knameCount.find(kName) == knameCount.end()) {
		knameCount[kName] = 0;
	} else {
		knameCount[kName] += 1;				
	}

	char numstr[21]; // enough to hold all numbers up to 64-bits
	sprintf(numstr, "%d", knameCount[kName]); // convert int to string

	if (INJ_DEBUG_LIGHT) {
		printf("%s: count=%d\n", kName.c_str(), knameCount[kName]);
	}

	ofs << kName << ":" << numstr;
	for (int i=0; i<NUM_INST_TYPES; i++) {
		ofs << ":" << injCountersInstType[i] ;
	}

	ofs << ":" << opWillNotExecuteCount; // print the number of operations that will not execute
	for (int i=0; i<SASSI_NUM_OPCODES; i++) {
		ofs << ":" << opCounters[i] ;
	}
	ofs << "\n";

#if TIMING
	gettimeofday(&end, NULL);

	long seconds, useconds;    
	seconds  = end.tv_sec  - start.tv_sec;
	useconds = end.tv_usec - start.tv_usec;
	float mTime = ((seconds) * 1000 + useconds/1000.0);
	printf("\nTime for %s:  %f ms\n", cbInfo->symbolName, mTime);
	mTotalTime += mTime;
#endif
} 

static void sassi_init() 
{
	if (INJ_DEBUG_LIGHT)
		printf("Writing to filename:%s\n", profileFilename.c_str());
	ofs.open(profileFilename.c_str(), std::ofstream::out);

	ofs << get_profile_format();
}

static void sassi_finalize(sassi::lazy_allocator::device_reset_reason reason)
{
	ofs.close();

#if TIMING
	printf("\nTotal kernel time: %f ms\n", mTotalTime);
#endif
}

static sassi::lazy_allocator profilerInit(sassi_init, sassi_finalize, onKernelEntry, onKernelExit); 
